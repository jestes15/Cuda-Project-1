#include "hip/hip_runtime.h"
﻿// Inclusion of header files for running CUDA in Visual Studio Pro 2019 (v142)
#include "hip/hip_runtime.h"
#include ""

// Inclusion of the required CUDA libriaries and header files
#include <hiprand.h>
#include <hip/hip_runtime.h>

// Inclusion of headers from the standard library in C
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>

#define CURAND_RNG_NON_DEFAULT 24

// Windows implementation of the Linux sys/time.h fnuctions needed in this program
#include <sys/timeb.h>
#include <sys/types.h>
#include <winsock2.h>

#define __need_clock_t
#include <time.h>

/* Structure describing CPU time used by a process and its children.  */
struct tms
{
    clock_t tms_utime;          /* User CPU time.  */
    clock_t tms_stime;          /* System CPU time.  */

    clock_t tms_cutime;         /* User CPU time of dead children.  */
    clock_t tms_cstime;         /* System CPU time of dead children.  */
};

// CUDA 8+ requirment
struct timezone {
    int tz_minuteswest; /* minutes west of Greenwich */
    int tz_dsttime; /* type of DST correction */
};

int gettimeofday(struct timeval* t, void* timezone)
{
    struct _timeb timebuffer;
    _ftime(&timebuffer);
    t->tv_sec = (long)timebuffer.time;
    t->tv_usec = 1000 * timebuffer.millitm;
    return 0;
}

/* Store the CPU time used by this process and all its
   dead children (and their dead children) in BUFFER.
   Return the elapsed real time, or (clock_t) -1 for errors.
   All times are in CLK_TCKths of a second.  */
clock_t times(struct tms* __buffer) {

    __buffer->tms_utime = clock();
    __buffer->tms_stime = 0;
    __buffer->tms_cstime = 0;
    __buffer->tms_cutime = 0;
    return __buffer->tms_utime;
}
typedef long long suseconds_t;

// CUDA error check to get error name
const char * CUDA_CHECK_VAL(hipError_t x) {
    size_t size = 50 * sizeof('a');
    const char* msg = (char*)malloc(size);

    switch (x) {
    case 0:
        msg = "hipSuccess";
    case 1:
        msg = "hipErrorInvalidValue";
    case 2:
        msg = "hipErrorOutOfMemory";
    case 3:
        msg = "hipErrorNotInitialized";
    case 4:
        msg = "hipErrorDeinitialized";
    case 5:
        msg = "hipErrorProfilerDisabled";
    case 9:
        msg = "hipErrorInvalidConfiguration";
    case 12:
        msg = "hipErrorInvalidPitchValue";
    case 13:
        msg = "hipErrorInvalidSymbol";
    case 18:
        msg = "hipErrorInvalidTexture";
    case 19:
        msg = "cudaErrorInvalidTextureBinding";
    case 20:
        msg = "hipErrorInvalidChannelDescriptor";
    case 21:
        msg = "hipErrorInvalidMemcpyDirection";
    case 26:
        msg = "cudaErrorInvalidFilterSetting";
    case 27:
        msg = "cudaErrorInvalidNormSetting";
    case 34:
        msg = "cudaErrorStubLibrary";
    case 35:
        msg = "hipErrorInsufficientDriver";
    case 36:
        msg = "cudaErrorCallRequiresNewerDriver";
    case 37:
        msg = "cudaErrorInvalidSurface";
    case 43:
        msg = "cudaErrorDuplicateVariableName";
    case 44:
        msg = "cudaErrorDuplicateTextureName";
    case 45:
        msg = "cudaErrorDuplicateSurfaceName";
    case 46:
        msg = "cudaErrorDevicesUnavailable";
    case 49:
        msg = "cudaErrorIncompatibleDriverContext";
    case 52:
        msg = "hipErrorMissingConfiguration";
    case 65:
        msg = "cudaErrorLaunchMaxDepthExceeded";
    case 66:
        msg = "cudaErrorLaunchFileScopedTex";
    case 67:
        msg = "cudaErrorLaunchFileScopedSurf";
    case 68:
        msg = "cudaErrorSyncDepthExceeded";
    case 69:
        msg = "cudaErrorLaunchPendingCountExceeded";
    case 98:
        msg = "hipErrorInvalidDeviceFunction";
    case 100:
        msg = "hipErrorNoDevice";
    case 101:
        msg = "hipErrorInvalidDevice";
    case 102:
        msg = "cudaErrorDeviceNotLicensed";
    case 103:
        msg = "cudaErrorSoftwareValidityNotEstablished";
    case 127:
        msg = "cudaErrorStartupFailure";
    case 200:
        msg = "hipErrorInvalidImage";
    case 201:
        msg = "hipErrorInvalidContext";
    case 205:
        msg = "hipErrorMapFailed";
    case 206:
        msg = "hipErrorUnmapFailed";
    case 207:
        msg = "hipErrorArrayIsMapped";
    case 208:
        msg = "hipErrorAlreadyMapped";
    case 209:
        msg = "hipErrorNoBinaryForGpu";
    case 210:
        msg = "hipErrorAlreadyAcquired";
    case 211:
        msg = "hipErrorNotMapped";
    case 212:
        msg = "hipErrorNotMappedAsArray";
    case 213:
        msg = "hipErrorNotMappedAsPointer";
    case 214:
        msg = "hipErrorECCNotCorrectable";
    case 215:
        msg = "hipErrorUnsupportedLimit";
    case 216:
        msg = "hipErrorContextAlreadyInUse";
    case 217:
        msg = "hipErrorPeerAccessUnsupported";
    case 218:
        msg = "hipErrorInvalidKernelFile";
    case 219:
        msg = "hipErrorInvalidGraphicsContext";
    case 220:
        msg = "cudaErrorNvlinkUncorrectable";
    case 221:
        msg = "cudaErrorJitCompilerNotFound";
    case 222:
        msg = "cudaErrorUnsupportedPtxVersion";
    case 223:
        msg = "cudaErrorJitCompilationDisabled";
    case 300:
        msg = "hipErrorInvalidSource";
    case 301:
        msg = "hipErrorFileNotFound";
    case 302:
        msg = "hipErrorSharedObjectSymbolNotFound";
    case 303:
        msg = "hipErrorSharedObjectInitFailed";
    case 304:
        msg = "hipErrorOperatingSystem";
    case 400:
        msg = "hipErrorInvalidHandle";
    case 401:
        msg = "hipErrorIllegalState";
    case 500:
        msg = "hipErrorNotFound";
    case 600:
        msg = "hipErrorNotReady";
    case 700:
        msg = "hipErrorIllegalAddress";
    case 701:
        msg = "hipErrorLaunchOutOfResources";
    case 702:
        msg = "hipErrorLaunchTimeOut";
    case 703:
        msg = "cudaErrorLaunchIncompatibleTexturing";
    case 704:
        msg = "hipErrorPeerAccessAlreadyEnabled";
    case 705:
        msg = "hipErrorPeerAccessNotEnabled";
    case 708:
        msg = "hipErrorSetOnActiveProcess";
    case 709:
        msg = "hipErrorContextIsDestroyed";
    case 710:
        msg = "hipErrorAssert";
    case 711:
        msg = "cudaErrorTooManyPeers";
    case 712:
        msg = "hipErrorHostMemoryAlreadyRegistered";
    case 713:
        msg = "hipErrorHostMemoryNotRegistered";
    case 714:
        msg = "cudaErrorHardwareStackError";
    case 715:
        msg = "cudaErrorIllegalInstruction";
    case 716:
        msg = "cudaErrorMisalignedAddress";
    case 717:
        msg = "cudaErrorInvalidAddressSpace";
    case 718:
        msg = "cudaErrorInvalidPc";
    case 719:
        msg = "hipErrorLaunchFailure";
    case 720:
        msg = "hipErrorCooperativeLaunchTooLarge";
    case 800:
        msg = "cudaErrorNotPermitted";
    case 801:
        msg = "hipErrorNotSupported";
    case 802:
        msg = "cudaErrorSystemNotReady";
    case 803:
        msg = "cudaErrorSystemDriverMismatch";
    case 804:
        msg = "cudaErrorCompatNotSupportedOnDevice";
    case 900:
        msg = "hipErrorStreamCaptureUnsupported";
    case 901:
        msg = "hipErrorStreamCaptureInvalidated";
    case 902:
        msg = "hipErrorStreamCaptureMerge";
    case 903:
        msg = "hipErrorStreamCaptureUnmatched";
    case 904:
        msg = "hipErrorStreamCaptureUnjoined";
    case 905:
        msg = "hipErrorStreamCaptureIsolation";
    case 906:
        msg = "hipErrorStreamCaptureImplicit";
    case 907:
        msg = "hipErrorCapturedEvent";
    case 908:
        msg = "hipErrorStreamCaptureWrongThread";
    case 909:
        msg = "cudaErrorTimeout";
    case 910:
        msg = "hipErrorGraphExecUpdateFailure";
    case 999:
        msg = "hipErrorUnknown";
    default:
        msg = "NonValidCudaError";
    }
    return msg;
}

// CURAND error check to get error name
const char * CURAND_CHECK_VAL(hiprandStatus_t x) {
    size_t size = 50 * sizeof('a');
    const char* msg = (char*)malloc(size);

    switch (x) {
    case 0:
        msg = "HIPRAND_STATUS_SUCCESS";
    case 100:
        msg = "HIPRAND_STATUS_VERSION_MISMATCH";
    case 101:
        msg = "HIPRAND_STATUS_NOT_INITIALIZED";
    case 102:
        msg = "HIPRAND_STATUS_ALLOCATION_FAILED";
    case 103:
        msg = "HIPRAND_STATUS_TYPE_ERROR";
    case 104:
        msg = "HIPRAND_STATUS_OUT_OF_RANGE";
    case 105:
        msg = "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
    case 106:
        msg = "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
    case 201:
        msg = "HIPRAND_STATUS_LAUNCH_FAILURE";
    case 202:
        msg = "HIPRAND_STATUS_PREEXISTING_FAILURE";
    case 203:
        msg = "HIPRAND_STATUS_INITIALIZATION_FAILED";
    case 204:
        msg = "HIPRAND_STATUS_ARCH_MISMATCH";
    case 999:
        msg = "HIPRAND_STATUS_INTERNAL_ERROR";
    default:
        msg = "NON_VALID_CURAND_ERROR";
    }
    return msg;
}

// Check method for checking the error status of a CUDA call
#define CUDA_CALL(x) { if(x != hipSuccess){ sprintf("Error: %s at %s:%d\n", CUDA_CHECK_VAL(x), __FILE__, __LINE__); return EXIT_FAILURE;}}

// Check method for checking the error status of a cuRAND call
#define CURAND_CALL(x) {if(x != HIPRAND_STATUS_SUCCESS){ sprintf("Error: %s at %s:%d\n", CURAND_CHECK_VAL(x), __FILE__, __LINE__); return EXIT_FAILURE;}}

// The kernel, which runs on the GPU when called
__global__ void kernel(int* a, int* b, int* c, size_t N)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) c[idx] = a[idx] * b[idx];
}

// Function for generating the same results as the GPU kernel, used for verification of results
__host__ void KernelCPUEd(int* a, int* b, int* c, size_t size)
{
    for (int i = 0; i < size; i++)
        c[i] = a[i] * b[i];
}

// Program to convert a float array to an integer array
__host__ void FtoIArray(int* dst, float* src, size_t nElem) {
    for (int i = 0; i < nElem; i++)
        dst[i] = (int)(src[i] * 1000);
}

// Function for verifying the array generated by the kernel is correct
__host__ bool inline CHECK(int* a, int* b, size_t size)
{
    double epsilon = 1.0E-8;
    for (int x = 0; x < size; x++)
    {
        if (a[x] - b[x] > epsilon)
            return true;
    }
    return false;
}

__host__ double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// Entry point to the program
int main(void) {
    size_t nElem = 1 << 24;
    size_t nBytes = nElem * sizeof(int);
    size_t nBytesF = nElem * sizeof(float);

    int* h_A, * h_B, * h_C, * GpuRef;
    int* d_A, * d_B, * d_C;

    float* devNumGen, * devNumGen2, * h_AR, * h_BR;

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CUDA_CALL(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CUDA_CALL(hipSetDevice(dev));

    hiprandGenerator_t gen, gen2;

    // Allocation of memory on the host for transferring data from host to device and vice versa
    h_A = (int*)malloc(nBytes);
    h_B = (int*)malloc(nBytes);
    h_C = (int*)malloc(nBytes);
    GpuRef = (int*)malloc(nBytes);

    // Allocation of memory on the device for storage of data needed by the kernel during runtime
    CUDA_CALL(hipMalloc((int**)&d_A, nBytes));
    CUDA_CALL(hipMalloc((int**)&d_B, nBytes));
    CUDA_CALL(hipMalloc((int**)&d_C, nBytes));

    // Allocation of memory on host and device for testing the CUDA number generator
    h_AR = (float*)malloc(nBytes);
    h_BR = (float*)malloc(nBytes);
    CUDA_CALL(hipMalloc((float**)&devNumGen, nBytesF));
    CUDA_CALL(hipMalloc((float**)&devNumGen2, nBytesF));

    // CUDA number generator function calls and return values
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandCreateGenerator(&gen2, HIPRAND_RNG_PSEUDO_DEFAULT));

    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen2, time(NULL) + 1));

    CURAND_CALL(hiprandGenerateUniform(gen, devNumGen, nElem));
    CURAND_CALL(hiprandGenerateUniform(gen2, devNumGen2, nElem));

    // Transfer random numbers generated on device to host
    CUDA_CALL(hipMemcpy(h_AR, devNumGen, nBytesF, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_BR, devNumGen2, nBytesF, hipMemcpyDeviceToHost));

    FtoIArray(h_A, h_AR, nElem);
    FtoIArray(h_B, h_BR, nElem);

    // Transfer of populated arrays to the device for use by the kernel
    CUDA_CALL(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    // Calculate block indices
    int iLen = 1024;
    dim3 block(iLen, 1);
    dim3 grid((nElem + block.x - 1) / block.x, 1);

    // Kernel call to run the calculation n the GPU, uses 1 block and nElem amount of threads in the block
    // Max threads in a block for RTX 2060 is 4096 threads
    double iStart = cpuSecond();
    kernel <<<grid, block>>> (d_A, d_B, d_C, nElem);
    CUDA_CALL(hipDeviceSynchronize());
    double iEnd = cpuSecond() - iStart;

    printf("Execution time of the GPU kernel <<<%d, %d>>>: %g\n", grid.x, block.x, iEnd);

    // Verification function that the kernel on the GPU is performing properly
    double iStartCPU = cpuSecond();
    KernelCPUEd(h_A, h_B, h_C, nElem);
    double iEndCPU = cpuSecond() - iStart;
    printf("Execution time of the CPU function %g\n", iEndCPU);

    // Transfer of data from Device to the host
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(GpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // Verification of data, compares data generated on the host to the data generated on the device
    // If the data is different, goto Exit is called and memory is freed, the the program ends
    if (CHECK(h_C, GpuRef, nElem))
    {
        printf("The arrays are not the same\n");
        goto Exit;
    }

Exit:
    // Destroy the cuRAND number generator
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CURAND_CALL(hiprandDestroyGenerator(gen2));

    //Free device memory
    CUDA_CALL(hipFree(d_A));
    CUDA_CALL(hipFree(d_B));
    CUDA_CALL(hipFree(d_C));
    CUDA_CALL(hipFree(devNumGen));
    CUDA_CALL(hipFree(devNumGen2));

    //Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(GpuRef);
    free(h_AR);
    free(h_BR);

    // Allows for the user to see the output when running in Visual Studio Pro 2019 (v142)
    char a;
    printf("Press Enter to continue");
    scanf("%c", &a);

    return 0;
}